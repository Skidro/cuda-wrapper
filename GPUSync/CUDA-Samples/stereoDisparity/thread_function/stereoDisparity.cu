#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* A CUDA program that demonstrates how to compute a stereo disparity map using
 *   SIMD SAD (Sum of Absolute Difference) intrinsics
 */

/*
 * The program's performance is dominated by the computation on the
 * execution engine (EE) while memory copies between Host and Device
 * using the copy engine (CE) are significantly less time consuming.
 *
 * This version uses a user allocated stream and asynchronous memory
 * copy operations (hipMemcpyAsync()).  Cuda kernel invocations on the
 * stream are also asynchronous.  hipStreamSynchronize() is used to 
 * synchronize with both the copy and kernel executions.  Host pinned
 * memory is not used because the copy operations are not a significant 
 * element of performance.
 *
 * The program depends on two input files containing the image 
 * representations for the left and right stereo images 
 * (stereo.im0.640x533.ppm and stereo.im1.640x533.ppm)
 * which must be in the directory with the executable.
 *
 * Modified by Don Smith, Department of Computer Science,
 * University of North Carolina at Chapel Hill
 * 2015
 */
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/types.h>
#include <unistd.h>
#include <sched.h>
#include <errno.h>

// includes, kernels
// For the CUDA runtime routines (prefixed with "cuda")
#include <hip/hip_runtime.h>
// the kernel code
#include "stereoDisparity_kernel.cuh"

// includes, project
#include <helper_functions.h>  // helper for shared that are common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper for checking cuda initialization and error checking
#include <helper_string.h>     // helper functions for string parsing

int iDivUp(int a, int b)
{
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

unsigned int numData;
dim3 numThreads;
dim3 numBlocks;
unsigned int *h_odata;
unsigned int *d_odata, *d_img0, *d_img1;
unsigned int memSize;
hipStream_t my_stream;
unsigned char *h_img0;
unsigned char *h_img1;
int minDisp;
int maxDisp;
unsigned int w, h;


void stereoDisparity() {
  //initalize the memory for output data to zeros
  for (unsigned int i = 0; i < numData; i++)
    h_odata[i] = 0;

  // copy host memory with images to device

  // this call is asynchronous so only the lock of CE can be handled in the wrapper
  checkCudaErrors(hipMemcpyAsync(d_img0,  h_img0, memSize, hipMemcpyHostToDevice, my_stream));

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(my_stream);

  // this call is asynchronous so only the lock of CE can be handled in the wrapper
  checkCudaErrors(hipMemcpyAsync(d_img1,  h_img1, memSize, hipMemcpyHostToDevice, my_stream));

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(my_stream);

  // copy host memory that was set to zero to initialize device output
  // this call is asynchronous so only the lock of CE can be handled in the wrapper
  checkCudaErrors(hipMemcpyAsync(d_odata, h_odata, memSize, hipMemcpyHostToDevice, my_stream));

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(my_stream);

  // First run the warmup kernel (which we'll use to get the GPU in the correct max power state)
  // lock of EE is handled in wrapper for hipLaunchByPtr()
  stereoDisparityKernel<<<numBlocks, numThreads, 0, my_stream>>>(d_img0, d_img1, d_odata, w, h, minDisp/2, maxDisp);

  // synchronize with the stream after kernel execution
  // the wrapper for this function releases any lock held (EE here)
  hipStreamSynchronize(my_stream);

  // copy host memory that was set to zero to initialize device output
  // this call is asynchronous so only the lock of CE can be handled in the wrapper
  checkCudaErrors(hipMemcpyAsync(d_odata, h_odata, memSize, hipMemcpyHostToDevice, my_stream));

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(my_stream);

  // launch the stereoDisparity kernel
  // lock of EE is handled in wrapper for hipLaunchByPtr()
  stereoDisparityKernel<<<numBlocks, numThreads, 0, my_stream>>>(d_img0, d_img1, d_odata, w, h, minDisp, maxDisp);

  // synchronize with the stream after kernel execution
  // the wrapper for this function releases any lock held (EE here)
  hipStreamSynchronize(my_stream);

  // Check to make sure the kernel didn't fail
  getLastCudaError("Kernel execution failed");

  //Copy result from device to host for verification
  // this call is asynchronous so only the lock of CE can be handled in the wrapper
  checkCudaErrors(hipMemcpyAsync(h_odata, d_odata, memSize, hipMemcpyDeviceToHost, my_stream));

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(my_stream);

#ifdef PRINT_CHECKSUM
  // calculate sum of resultant GPU image
  // This verification is applied only to the
  // last result computed
  unsigned int checkSum = 0;
  for (unsigned int i=0 ; i <w *h ; i++) {
    checkSum += h_odata[i];
  }
  if (checkSum == 4293895789) //valid checksum only for these two images
    printf("Test PASSED\n");
  else {
    fprintf(stderr, "Verification failed, GPU Checksum = %u, ", checkSum);
    exit(-1);
  }
#endif

#ifdef WRITE_DISPARITY
  // write out the resulting disparity image.
  // creates file in directory containing executable
  unsigned char *dispOut = (unsigned char *)malloc(numData);
  int mult = 20;

  char fnameOut[50] = "";
  strcat(fnameOut, "output_GPU.pgm");

  for (unsigned int i=0; i<numData; i++)
  {
    dispOut[i] = (int)h_odata[i]*mult;
  }

  printf("GPU image: <%s>\n", fnameOut);
  sdkSavePGM(fnameOut, dispOut, w, h);
  if (dispOut != NULL) free(dispOut);
#endif
  // prepare to clean up 
  // wrapper will release any lock held
  hipDeviceSynchronize();

  // cleanup device memory
  checkCudaErrors(hipFree(d_odata));
  checkCudaErrors(hipFree(d_img0));
  checkCudaErrors(hipFree(d_img1));

  // cleanup host memory
  if (h_odata != NULL) free(h_odata);

  if (h_img0 != NULL) free(h_img0);

  if (h_img1 != NULL) free(h_img1);

  // finish clean up with deleting the user-created stream
  hipStreamDestroy(my_stream);

  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  hipDeviceReset();
}

int main(int argc, char **argv)
{
  int sync_level = 2; //default -- process blocking

  /*
   * The only parameter is an integer that indicates the desired level of
   * synchronization used by the GPU driver (values defined below).  The
   * specified level is used in hipSetDeviceFlags() to set the level
   * prior to initialization.
   */
  if (argc == 2)
    sync_level = atoi(argv[1]);
  // level 0 - spin polling (busy waiting) for GPU to finish
  // level 1 - yield each time through the polling loop to let another thread run
  // level 2 - block process waiting for GPU to finish
  switch (sync_level)
  {
    case 0:
      hipSetDeviceFlags(hipDeviceScheduleSpin);
      break;
    case 1:
      hipSetDeviceFlags(hipDeviceScheduleYield);
      break;
    default:
      hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  }

  // follow convention and initialize CUDA/GPU
  // used here to invoke initialization of GPU locking
  hipFree(0);

  // use device 0, the only one on a TK1
  hipSetDevice(0);

  // create a user-defined stream
  hipStreamCreate(&my_stream);

  // Search paramters
  minDisp = -16;
  maxDisp = 0;

  // Load image data
  // functions allocate memory for the images on host side
  // initialize pointers to NULL to request lib call to allocate as needed
  // PPM images are loaded into 4 byte/pixel memory (RGBX)
  h_img0 = NULL;
  h_img1 = NULL;
  char *fname0 = sdkFindFilePath("stereo.im0.640x533.ppm", argv[0]);
  char *fname1 = sdkFindFilePath("stereo.im1.640x533.ppm", argv[0]);

  if (!sdkLoadPPM4ub(fname0, &h_img0, &w, &h))
  {
    fprintf(stderr, "Failed to load <%s>\n", fname0);
    exit(-1);
  }

  if (!sdkLoadPPM4ub(fname1, &h_img1, &w, &h))
  {
    fprintf(stderr, "Failed to load <%s>\n", fname1);
    exit(-1);
  }

  // set up parameters used in the rest of program
  numThreads = dim3(blockSize_x, blockSize_y, 1);
  numBlocks = dim3(iDivUp(w, numThreads.x), iDivUp(h, numThreads.y));
  numData = w*h;
  memSize = sizeof(int) * numData;

  //allocate memory for the result on host side
  h_odata = (unsigned int *)malloc(memSize);

  // allocate device memory for inputs and result
  checkCudaErrors(hipMalloc((void **) &d_odata, memSize));
  checkCudaErrors(hipMalloc((void **) &d_img0, memSize));
  checkCudaErrors(hipMalloc((void **) &d_img1, memSize));

  // more setup for using the GPU
  size_t offset = 0;
  hipChannelFormatDesc ca_desc0 = hipCreateChannelDesc<unsigned int>();
  hipChannelFormatDesc ca_desc1 = hipCreateChannelDesc<unsigned int>();

  tex2Dleft.addressMode[0] = hipAddressModeClamp;
  tex2Dleft.addressMode[1] = hipAddressModeClamp;
  tex2Dleft.filterMode     = hipFilterModePoint;
  tex2Dleft.normalized     = false;
  tex2Dright.addressMode[0] = hipAddressModeClamp;
  tex2Dright.addressMode[1] = hipAddressModeClamp;
  tex2Dright.filterMode     = hipFilterModePoint;
  tex2Dright.normalized     = false;
  checkCudaErrors(hipBindTexture2D(&offset, tex2Dleft,  d_img0, ca_desc0, w, h, w*4));
  assert(offset == 0);

  checkCudaErrors(hipBindTexture2D(&offset, tex2Dright, d_img1, ca_desc1, w, h, w*4));
  assert(offset == 0);

  // all setup and initialization complete, start iterations
  stereoDisparity();
}
