#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* A CUDA program that demonstrates how to compute a stereo disparity map using
 *   SIMD SAD (Sum of Absolute Difference) intrinsics
 */

/*
 * Modified to iterate the stereo disparity multiple times to use as
 * benchmark/stress test for GPU locking using Cuda call-wrapping
 * functions.  The program's performance is dominated by 
 * the computation on the execution engine (EE) while memory copies 
 * between Host and Device using the copy engine (CE) are significantly
 * less time consuming.
 *
 * This version uses the default stream and synchronous memory copy
 * operations (hipMemcpy()).  Cuda kernel invocations are always
 * asynchronous so hipDeviceSynchronize() is used to synchronize
 * with kernel execution.  Host pinned memory is not used because
 * the copy operations are not a significant element of performance.
 *
 * The program depends on two input files containing the image 
 * representations for the left and right stereo images 
 * (stereo.im0.640x533.ppm and stereo.im1.640x533.ppm)
 * which must be in the directory with the executable.
 *
 * Modified by Don Smith, Department of Computer Science,
 * University of North Carolina at Chapel Hill
 * 2015
 */
// control number of iterations by count or elapsed time
#define MAX_LOOPS 10000 // iteration count
#define TIME_LENGTH 30  // elapsed time (seconds)

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/types.h>
#include <unistd.h>
#include <sched.h>
#include <errno.h>

// includes, kernels
// For the CUDA runtime routines (prefixed with "cuda")
#include <hip/hip_runtime.h>
// The kernel code
#include "stereoDisparity_kernel.cuh"

// includes, project
#include <helper_functions.h>  // helper for shared that are common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper for checking cuda initialization and error checking
#include <helper_string.h>     // helper functions for string parsing

int iDivUp(int a, int b)
{
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
  runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! CUDA Sample for calculating depth maps
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
  hipDeviceProp_t deviceProp;
  deviceProp.major = 0;
  deviceProp.minor = 0;
  int i;
  int count = 0;

  int sync_level = 2; //default -- process blocking

  pid_t my_pid;
  time_t start_time, now, elapsed;

  my_pid = getpid();

  /*
   * The only parameter is an integer that indicates the desired level of
   * synchronization used by the GPU driver (values defined below).  The
   * specified level is used in hipSetDeviceFlags() to set the level
   * prior to initialization.
   */
  if (argc == 2)
    sync_level = atoi(argv[1]);
  // level 0 - spin polling (busy waiting) for GPU to finish
  // level 1 - yield each time through the polling loop to let another thread run
  // level 2 - block process waiting for GPU to finish
  switch (sync_level)
  {
    case 0:
      hipSetDeviceFlags(hipDeviceScheduleSpin);
      printf("PID %d started > Synch Level is Spin\n", my_pid);
      break;
    case 1:
      hipSetDeviceFlags(hipDeviceScheduleYield);
      printf("PID %d started > Synch Level is Yield\n", my_pid);
      break;
    default:
      hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
      printf("PID %d started > Synch Level is Block\n", my_pid);
  }

  // follow convention and initialize CUDA/GPU
  // used here to invoke initialization of GPU locking
  hipFree(0);

  // use device 0, the only one on a TK1
  hipSetDevice(0);

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, 0));

  // Search paramters
  int minDisp = -16;
  int maxDisp = 0;

  // Load image data
  // functions allocate memory for the images on host side
  // initialize pointers to NULL to request lib call to allocate as needed
  // PPM images are loaded into 4 byte/pixel memory (RGBX)
  unsigned char *h_img0 = NULL;
  unsigned char *h_img1 = NULL;
  unsigned int w, h;
  char *fname0 = sdkFindFilePath("stereo.im0.640x533.ppm", argv[0]);
  char *fname1 = sdkFindFilePath("stereo.im1.640x533.ppm", argv[0]);

  if (!sdkLoadPPM4ub(fname0, &h_img0, &w, &h))
  {
    fprintf(stderr, "PID %d Failed to load <%s>\n", my_pid, fname0);
    exit(-1);
  }

  if (!sdkLoadPPM4ub(fname1, &h_img1, &w, &h))
  {
    fprintf(stderr, "PID %d Failed to load <%s>\n", my_pid, fname1);
    exit(-1);
  }

  // set up parameters used in rest of program
  dim3 numThreads = dim3(blockSize_x, blockSize_y, 1);
  dim3 numBlocks = dim3(iDivUp(w, numThreads.x), iDivUp(h, numThreads.y));
  unsigned int numData = w*h;
  unsigned int memSize = sizeof(int) * numData;

  //allocate memory for the result on host side
  unsigned int *h_odata = (unsigned int *)malloc(memSize);

  // allocate device memory for inputs and the result
  unsigned int *d_odata, *d_img0, *d_img1;
  checkCudaErrors(hipMalloc((void **) &d_odata, memSize));
  checkCudaErrors(hipMalloc((void **) &d_img0, memSize));
  checkCudaErrors(hipMalloc((void **) &d_img1, memSize));

  // more setup for using the GPU
  size_t offset = 0;
  hipChannelFormatDesc ca_desc0 = hipCreateChannelDesc<unsigned int>();
  hipChannelFormatDesc ca_desc1 = hipCreateChannelDesc<unsigned int>();

  tex2Dleft.addressMode[0] = hipAddressModeClamp;
  tex2Dleft.addressMode[1] = hipAddressModeClamp;
  tex2Dleft.filterMode     = hipFilterModePoint;
  tex2Dleft.normalized     = false;
  tex2Dright.addressMode[0] = hipAddressModeClamp;
  tex2Dright.addressMode[1] = hipAddressModeClamp;
  tex2Dright.filterMode     = hipFilterModePoint;
  tex2Dright.normalized     = false;
  checkCudaErrors(hipBindTexture2D(&offset, tex2Dleft,  d_img0, ca_desc0, w, h, w*4));
  assert(offset == 0);

  checkCudaErrors(hipBindTexture2D(&offset, tex2Dright, d_img1, ca_desc1, w, h, w*4));
  assert(offset == 0);
  // all setup and initialization complete, start iterations


  printf("PID %d Iterating stereoDisparity CUDA Kernel for %d seconds, %d max loops\n", my_pid, TIME_LENGTH, MAX_LOOPS);
  now = start_time = time(NULL);

  for (i = 0; 
      ((now - TIME_LENGTH) < start_time) &&
      i < MAX_LOOPS; i++) {

    //initalize the memory for output data to zeros
    for (unsigned int i = 0; i < numData; i++)
      h_odata[i] = 0;

    // copy host memory with images to device
    // copy host memory that was set to zero to initialize device output
    // these calls are synchronous so lock/unlock of CE can be handled in wrappers
    checkCudaErrors(hipMemcpy(d_img0,  h_img0, memSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_img1,  h_img1, memSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_odata, h_odata, memSize, hipMemcpyHostToDevice));

    // First run the warmup kernel (which we'll use to get the GPU in the correct max power state)
    // lock of EE is handled in wrapper for hipLaunchByPtr()
    stereoDisparityKernel<<<numBlocks, numThreads>>>(d_img0, d_img1, d_odata, w, h, minDisp/2, maxDisp);

    // synchronize the default stream
    // used here so wrapper function can release EE lock
    hipDeviceSynchronize();

    // copy host memory that was set to zero to initialize device output
    checkCudaErrors(hipMemcpy(d_odata, h_odata, memSize, hipMemcpyHostToDevice));

    // launch the stereoDisparity kernel
    // lock of EE is handled in wrapper for hipLaunchByPtr()
    stereoDisparityKernel<<<numBlocks, numThreads>>>(d_img0, d_img1, d_odata, w, h, minDisp, maxDisp);

    // synchronize the default stream
    // used here so wrapper function can release EE lock
    hipDeviceSynchronize();

    // Check to make sure the kernel didn't fail
    getLastCudaError("Kernel execution failed");

    //Copy result from device to host for verification
    // these calls are synchronous so lock/unlock of CE can be handled in wrappers
    checkCudaErrors(hipMemcpy(h_odata, d_odata, memSize, hipMemcpyDeviceToHost));

    now = time(NULL);

  } // ends for loop
  elapsed = now - start_time;
  count = i;

  // calculate checksum of resultant GPU image
  // This verification is applied only to the 
  // last result computed
  unsigned int checkSum = 0;

  for (unsigned int i=0 ; i<w *h ; i++)
  {
    checkSum += h_odata[i];
  }

  if (checkSum == 4293895789) //valid checksum only for these two images
    printf("PID %d Test PASSED\n", my_pid);
  else {
    fprintf(stderr, "PID %d verification failed, GPU Checksum = %u, ", my_pid, checkSum);
    exit(-1);
  }

  printf("PID %d completed %d, duration %ld seconds\n", my_pid, count, elapsed);

#ifdef WRITE_DISPARITY
  // write out the resulting disparity image.
  // creates file in directory containing executable
  unsigned char *dispOut = (unsigned char *)malloc(numData);
  int mult = 20;

  char fnameOut[50] = "";
  sprintf(fnameOut,"PID_%d_", my_pid);
  strcat(fnameOut, "output_GPU.pgm");

  for (unsigned int i=0; i<numData; i++)
  {
    dispOut[i] = (int)h_odata[i]*mult;
  }

  printf("GPU image: <%s>\n", fnameOut);
  sdkSavePGM(fnameOut, dispOut, w, h);
  if (dispOut != NULL) free(dispOut);

#endif

  // cleanup device memory
  checkCudaErrors(hipFree(d_odata));
  checkCudaErrors(hipFree(d_img0));
  checkCudaErrors(hipFree(d_img1));

  // cleanup host memory
  if (h_odata != NULL) free(h_odata);

  if (h_img0 != NULL) free(h_img0);

  if (h_img1 != NULL) free(h_img1);

  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  hipDeviceReset();
}
